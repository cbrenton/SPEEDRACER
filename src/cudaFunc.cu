#include "hip/hip_runtime.h"
#include "cudaFunc.h"

//using namespace std; 
/*file for implementing the cuda version of rasterization functions*/
//

void cudasafe( hipError_t error, char* message)
{
if(error!=hipSuccess) { fprintf(stderr,"ERROR: %s : %i\n",message,error); exit(-1); }
}
tri_t* sendTrianglesToDevice(tri_t* triList,int size)
{
   tri_t* tri_d;
   cudasafe(hipMalloc(&tri_d,sizeof(tri_t)*size),"tri send malloc");
   cudasafe(hipMemcpy(tri_d,triList,sizeof(tri_t)*size,hipMemcpyHostToDevice),"tri cpy");
   //hipMemcpyToSymbol(HIP_SYMBOL(tri_d),triList,sizeof(tri_t)*size);
   //printf("TEST\n");
   //printf("%d %d %d \n",tri_d->pt[0],tri_d->pt[1],tri_d->pt[2]);
   return tri_d;
}
tri_t* testTriangles(tri_t* input,int size)
{
   tri_t* tri_d;
   tri_d = sendTrianglesToDevice(input,size);
   return retrieveTrianglesFromDevice(tri_d,size);
}
//function for writing the converted points onto the graphics card
point_t* sendPointToDeviceConst(point_t* pointList,int size)
{
   point_t* point_d;
   cudasafe(hipMalloc(&point_d,sizeof(point_t)*size),"point malloc");
   cudasafe(hipMemcpy(point_d,pointList,sizeof(point_t)*size,hipMemcpyHostToDevice),"point memcpy");
   //hipMemcpyToSymbol(HIP_SYMBOL(point_d),pointList,sizeof(point_t)*size);
   return point_d;
}
point_t* testPoints(point_t* input,int size)
{
   point_t* point_d;
   point_d = sendPointsToDevice(input,size);
   return retrievePointsFromDevice(point_d,size);
}


//Function for retrieving the converted tri_t after the kernel has been run
//assumes that the tri_return points to a malloced pointer for the given size
tri_t* retrieveTrianglesFromDevice(tri_t* tri_d,int size)
{
   tri_t* tri_r;
   tri_r = (tri_t*)malloc(sizeof(tri_t)*size);
   cudasafe(hipMemcpy(tri_r,tri_d,sizeof(tri_t)*size,hipMemcpyDeviceToHost),"retieve Tri");
   cudasafe(hipFree(tri_d),"free tri");
   return tri_r;
}
point_t* retrievePointsFromDevice(point_t* point_d,int size)
{
   point_t* point_r;
   point_r = (point_t*)malloc(sizeof(point_t)*size);
   cudasafe(hipMemcpy(point_r,point_d,sizeof(point_t)*size,hipMemcpyDeviceToHost),"retrieve points");
   cudasafe(hipFree(point_d),"points");
   return point_r;

}

//function for sending the points to the device, returns a pointer to the mem
point_t* sendPointsToDevice(point_t* pointList,int size)
{
   point_t* point_d;
   cudasafe(hipMalloc(&point_d,sizeof(point_t)*size),"malloc point");
   cudasafe(hipMemcpy(point_d,pointList,sizeof(point_t)*size,hipMemcpyHostToDevice),"cpy point");
   return point_d;

}

vec3_t* sendColorToDevice(colorbuffer* colorbuff,int size)
{
   vec3_t* color_d;
   //printf("send color size %d\n",size);
   cudasafe(hipMalloc(&color_d,sizeof(vec3_t)*size),"color");
   cudasafe(hipMemcpy(color_d,colorbuff->data,sizeof(vec3_t)*size,hipMemcpyHostToDevice),"color2");
   return color_d;

}
vec3_t* testColor(colorbuffer* input,int size)
{
   vec3_t* data;
   cudasafe(hipMalloc(&data,sizeof(vec3_t)*size),"test colorM");
   cudasafe(hipMemcpy(data,input,sizeof(vec3_t)*size,hipMemcpyHostToDevice),"memcpy test");
   return retrieveColorFromDevice(data,size);

}
//function to retrieve color from the graphics card
vec3_t* retrieveColorFromDevice(vec3_t* color_d,int size)
{
   vec3_t* color_r;
   color_r = (vec3_t*)malloc(sizeof(vec3_t)*size);
   cudasafe(hipMemcpy(color_r,color_d,sizeof(vec3_t)*size,hipMemcpyDeviceToHost),"color ret");
   return color_r;

}


//function to run the entire convert process
point_t* cudaConvertCoords(point_t* pointList,int size, int h, int w,vec_t scale)
{
   point_t* point_d;// pointer for holding the result of calc
   dim3 dimBlock(h/50 +1, w/50 +1);
   dim3 dimGrid(50,3);
   point_d= sendPointsToDevice(pointList,size);
   cudaCoordinateCalc<<<dimBlock,dimGrid>>>(point_d, size, h, w,scale);
   return retrievePointsFromDevice(point_d,size);
   
}
//function to send the zbuffer to the device
vec_t* sendZBufferToDevice(zbuffer* zbuff,int size)
{
   vec_t* zbuff_d;
   cudasafe(hipMalloc(&zbuff_d,sizeof(vec_t)*size),"send zbuff");
   cudasafe(hipMemcpy(zbuff_d,zbuff->data,sizeof(vec_t)*size,hipMemcpyHostToDevice),"zbuff cpy");
   return zbuff_d;

}



//kernel for finding the new points after conversion 
//****confirmed works***
__global__ void cudaCoordinateCalc(point_t* point_d, int listSize,int w_in, int h_in,vec_t scale )
{
   int location;
   location =blockIdx.x*50 +threadIdx.x;
   //checki if this thread is within range
   if(location >= listSize)
   {
      return;
   }
   scale = (vec_t) 1.00;
   int tpX,tpY; 
   int h= h_in;
   int w= w_in;
   vec_t dim = scale;
   // Convert x.
   vec_t tmpX = point_d[location].coords.v[0] + dim; // Shift.
   tpX = (int)(tmpX * (vec_t)(w - 1) / 2 * ((vec_t)1.0 / dim)); // Scale.
   // Convert y.
   vec_t tmpY = point_d[location].coords.v[1] + dim; // Shift. 
   tpY = (int)(tmpY * (vec_t)(h - 1) / 2 * ((vec_t)1.0 / dim)); // Scale.
   point_d[location].pX = tpX;
   point_d[location].pY = tpY;
   return;
}

//function for setting up the lock on the zbuffer and color buffer for use with semaphores 
int* setupBuffLock(int size)
{
    int* lock;
   cudasafe(hipMalloc(&lock,sizeof(int)*size),"lock array");
   return lock;
}

//function for rasterization,called by main
//HAS NOT BEEN TESTED********
void cudaRasterize(tri_t* tri,int tri_size,point_t* points,int ptSize,colorbuffer* cbuff,
   zbuffer* zbuff)
{
   vec3_t* color_d;//device color buffer
   int*lock;
   tri_t* tri_d;
   tri_t* temp;
   point_t* point_d;
   vec3_t* temp_c;
   point_t* temp_p;


   //temp = testTriangles(tri+9,tri_size);
  // printf("passed %d %d %d\n",temp->pt0,temp->pt1,temp->pt2);
   //printf("sanity check\n");
   //temp_p = testPoints(points,ptSize);
   //printf("%f %f %f \n",temp_p->coords.v[0],temp_p->coords.v[1],temp_p->coords.v[2]);


   point_d = sendPointsToDevice(points,ptSize);
   tri_d = sendTrianglesToDevice(tri,tri_size);
   
   int buffsize = cbuff->w * cbuff->h;//calculate the buffer size
   
   //printf("buff size %d\n",buffsize);
   
   color_d = sendColorToDevice(cbuff,buffsize);//setup the color buffer on device
  
  // vec3_t* test_c;
  // test_c = testColor(cbuff,buffsize);
  // printf("%f,%f,%f\n",test_c[0].v[0],test_c[0].v[1],test_c[0].v[2]);
  
   vec_t* zbuff_d;
   zbuff_d = sendZBufferToDevice(zbuff,buffsize);
   
   //Kernel size setup
   dim3 dimBlock(tri_size/20 +1);
   dim3 dimGrid(20,1);
   
   lock= setupBuffLock(buffsize);
   
   //Call the cuda Kernel
   //printf("just before the kernel\n");
   cudaRasterizeKernel<<<dimBlock,dimGrid>>>(tri_d,tri_size,point_d,color_d,zbuff_d,cbuff->h,lock);
   
   cbuff->data = retrieveColorFromDevice(color_d,buffsize);
   //printf("just after the kernel\n");
   return;

}

//function run in each thread to rasterize with the given data
__global__ void cudaRasterizeKernel(tri_t* tri_d,int tri_size,point_t* point_d,vec3_t* color_d,
   vec_t* zbuff_d,int height,int* lock)
{
   
   //check if this thread is within range of applicable triangles
   if(tri_size < blockIdx.x *20 +threadIdx.x)
   {
      //printf("INVALID THREAD\n");
      return;
   }
   
   tri_t *tri = &tri_d[blockIdx.x*20+threadIdx.x];//register for the current triangle value


  // for (int x = tri->extents[0]; x < tri->extents[1]; x++)
   for(int x=0;x<60;x++)
   {
    //  for (int y = tri->extents[2]; y < tri->extents[3]; y++)
      for(int y=0;y<60;y++)
      {
         
         vec_t z = zbuff_d[x * height + y];
         vec_t t = FLT_MAX;
         vec_t bary[3];
        
         //printf("just before hit\n");
         // int tmpi = tri_d[blockIdx.x*20+threadIdx.x].pt0;
         //printf("pt[0]: %d\n", tmpi);
         
         //Check if hit, then write
         if (cudaHit(&tri_d[blockIdx.x*20+threadIdx.x],point_d,x,y,&t,bary))
         {
            // printf("hit pass\n");
            
            // Check the z-buffer to see if this should be written.
            if (t > z)
            {
             
               /*
               // Calculate the normal.
               vec_t normal[3] = {
                  tri->normal[0],
                  tri->normal[1],
                  tri->normal[2]
               };
               // Calculate the color (N dot L).
               vec_t colorMag = dot_d(normal, light);
               */
 
               vec_t colorMag = 1.f;
               if (colorMag < 0)
               {
                  colorMag *= -1.f;
               }
               // Clamp the color to (0.0, 1.0).
               colorMag = max((vec_t)0.f, min(colorMag, (vec_t)1.f));
               
              // Write to color buffer
              __syncthreads();
    
              //printf("after sync \n");

              if(atomicMin_f((float*)&zbuff_d[x * height + y],z) == z)//check if current min
              {
                while(atomicAdd((float*)&lock[x * height + y],1.f) == 0)//check if current pos is open
                { 
                   //write to the buffer, commented out for testing
                  /* color_d[x * height + y].v[0] = bary[0];
                   color_d[x * height + y].v[1] = bary[1];
                   color_d[x * height + y].v[2] = bary[2];*/
                   printf("color\n"); 
                   atomicExch(&lock[x * height + y],0);
                   break;
                  //printf("TEST53\n");
                }
              }
               // Write to z-buffer.
               // *z = t;
            }
         }
      }
   }
}

//function to subs for missing atomic min on floats
__device__ float atomicMin_f(float* val,float z)
{
   float temp = atomicExch(val,z);
   if(z<temp)//case where z was the min
   {
      return z;
   }
   else
   {
      atomicExch(val,temp);
      return temp;
   }
}

//function to print the matrix
__device__ void printMat(vec_t *m)
{
printf("{%f %f %f\n%f %f %f\n%f %f %f}\n", m[0], m[1], m[2],
   m[3], m[4], m[5],
   m[6], m[7], m[8]);
}

/*__device__ bool cudaHit(int x, int y, vec_t *t, vec_t *bary,tri_t* tri_d,point_t* point_d,int index*/
__device__ bool cudaHit(tri_t* tri, point_t *ptList, int x, int y, vec_t *t, vec_t *bary)
{
   return false;
   /*
   //printf("hit started\n");
   //if (x < tri->extents[0] || x > tri->extents[1] ||
    //     y < tri->extents[2] || y > tri->extents[3])
     // return false;
   //printf("passed the test\n");
   bool hit = true;

   vec_t bBeta, bGamma, bT;

   vec_t pix[3] = {(vec_t)x, (vec_t)y, 0.f};
    vec_t screenPt[9];
   //printf("cudahit inti\n");
   //printf("pt[0]: %d\n", tri->pt0);

//printf("TEST1\n");
   screenPt[0] = (vec_t)ptList[tri->pt0].pX;
   screenPt[1] = (vec_t)ptList[tri->pt0].pY;
   screenPt[2] = 0.f;
   screenPt[3] = (vec_t)ptList[tri->pt1].pX;
   screenPt[4] = (vec_t)ptList[tri->pt1].pY;
   screenPt[5] = 0.f;
   screenPt[6] = (vec_t)ptList[tri->pt2].pX;
   screenPt[7] = (vec_t)ptList[tri->pt2].pY;
   screenPt[8] = 0.f;
//printf("screen %d\n",ptList[0].pX);

   vec_t A[9] = {screenPt[0 * 3 + 0], screenPt[1 * 3 + 0], screenPt[2 * 3 + 0],
      screenPt[0 * 3 + 1], screenPt[1 * 3 + 1], screenPt[2 * 3 + 1],
      1.f, 1.f, 1.f};

   vec_t detA = det_d(A);
//printMat(A);
//printf("oh yeeeeaaaahhh\n");
//printf("%d\n", detA);

   if (detA == 0)
   {
      return false;
   }
//printf("past detA test\n");
   vec_t baryT[9] = {pix[0], screenPt[1 * 3 + 0], screenPt[2 * 3 + 0],
      pix[1], screenPt[1 * 3 + 1], screenPt[2 * 3 + 1],
      1.f, 1.f, 1.f};

   bT = det_d(baryT) / detA;

   if (bT < 0)
   {
      hit = false;
   }
   else
   {
      vec_t baryGamma[9] = {screenPt[0 * 3 + 0], pix[0], screenPt[2 * 3 + 0],
         screenPt[0 * 3 + 1], pix[1], screenPt[2 * 3 + 1],
         1.f, 1.f, 1.f};

      bGamma = det_d(baryGamma) / detA;

      if (bGamma < 0 || bGamma > 1)
      {
         hit = false;
      }
      else
      {
         vec_t baryBeta[9] = {screenPt[0 * 3 + 0], screenPt[1 * 3 + 0], pix[0],
            screenPt[0 * 3 + 1], screenPt[1 * 3 + 1], pix[1],
            1.f, 1.f, 1.f};

         bBeta = det_d(baryBeta) / detA;

         if (bBeta < 0 || bBeta > 1 - bGamma)
         {
            hit = false;
         }
      }
   }

   if (hit)
   {
      *t = bT * ptList[tri->pt0].coords.v[2] + bBeta * ptList[tri->pt1].coords.v[2] + bGamma *
         ptList[tri->pt2].coords.v[2];
      if (bary)
      {
         bary[0] = bT;
         bary[1] = bBeta;
         bary[2] = bGamma;
      }
   }
   printf("HIT!!! %d %d\n",x,y);
   return hit;
   */
}


//function for dot product
__device__ vec_t dot_d(vec_t* a, vec_t* b)
{
   return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

//function for determinate
__device__ vec_t det_d(vec_t* data)
{
   return data[0 * 3 + 0] * data[1 * 3 + 1] * data[2 * 3 + 2] + data[0 * 3 + 1] * data[1 * 3 + 2] *
      data[2 * 3 + 0] + data[0 * 3 + 2] * data[1 * 3 + 0] * data[2 * 3 + 1] - data[0 * 3 + 2] *
      data[1 * 3 + 1] * data[2 * 3 + 0] - data[0 * 3 + 0] * data[1 * 3 + 2] * data[2 * 3 + 1] -
      data[0 * 3 + 1] * data[1 * 3 + 0] * data[2 * 3 + 2];
}
